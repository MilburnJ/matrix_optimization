#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_fp16.h> // For half-precision support
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Utility to check CUDA errors
#define CUDA_CHECK(call)                                                      \
    {                                                                         \
        const hipError_t error = call;                                       \
        if (error != hipSuccess) {                                           \
            std::cerr << "CUDA Error: " << __FILE__ << ":" << __LINE__        \
                      << ", " << hipGetErrorString(error) << std::endl;      \
            exit(1);                                                          \
        }                                                                     \
    }

// Utility to check cuBLAS errors
#define CUBLAS_CHECK(call)                                                    \
    {                                                                         \
        const hipblasStatus_t status = call;                                   \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                \
            std::cerr << "cuBLAS Error: " << __FILE__ << ":" << __LINE__      \
                      << ", Status Code: " << status << std::endl;            \
            exit(1);                                                          \
        }                                                                     \
    }

int main() {
    int n = 1000; // Matrix dimensions (1000x1000)

    // Track overall execution time
    auto total_start = std::chrono::high_resolution_clock::now();

    // Allocate and initialize host memory in FP16
    std::vector<half> h_A(n * n, __float2half(1.0f)); // Initialize A with 1.0
    std::vector<half> h_B(n * n, __float2half(1.0f)); // Initialize B with 1.0
    std::vector<half> h_C(n * n, __float2half(0.0f)); // Initialize C with 0.0

    // Allocate device memory
    half *d_A, *d_B, *d_C, *d_intermediate;
    CUDA_CHECK(hipMalloc(&d_A, n * n * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_B, n * n * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_C, n * n * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_intermediate, n * n * sizeof(half)));

    // Copy matrices A and B to the device
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), n * n * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B.data(), n * n * sizeof(half), hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Enable Tensor Core acceleration
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    // Perform first matrix multiplication: d_intermediate = d_A * d_B
    half alpha = __float2half(1.0f);
    half beta = __float2half(0.0f);
    CUBLAS_CHECK(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N, // No transpose for A and B
        n, n, n,                 // Matrix dimensions
        &alpha,
        d_A, HIP_R_16F, n,
        d_B, HIP_R_16F, n,
        &beta,
        d_intermediate, HIP_R_16F, n,
        HIP_R_32F, // Compute in FP32
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    ));

    // Perform second matrix multiplication: d_C = d_intermediate * d_B
    CUBLAS_CHECK(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        d_intermediate, HIP_R_16F, n,
        d_B, HIP_R_16F, n,
        &beta,
        d_C, HIP_R_16F, n,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    ));

    // Copy the result back to host
    CUDA_CHECK(hipMemcpy(h_C.data(), d_C, n * n * sizeof(half), hipMemcpyDeviceToHost));

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_intermediate);
    hipblasDestroy(handle);

    // Track total execution time
    auto total_end = std::chrono::high_resolution_clock::now();
    std::cout << "Total execution time: "
              << std::chrono::duration<double>(total_end - total_start).count()
              << " seconds" << std::endl;

    return 0;
}
