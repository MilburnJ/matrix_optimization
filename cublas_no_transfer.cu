#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Utility to check CUDA errors
#define CUDA_CHECK(call)                                                      \
    {                                                                         \
        const hipError_t error = call;                                       \
        if (error != hipSuccess) {                                           \
            std::cerr << "CUDA Error: " << __FILE__ << ":" << __LINE__        \
                      << ", " << hipGetErrorString(error) << std::endl;      \
            exit(1);                                                          \
        }                                                                     \
    }

// Utility to check cuBLAS errors
#define CUBLAS_CHECK(call)                                                    \
    {                                                                         \
        const hipblasStatus_t status = call;                                   \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                \
            std::cerr << "cuBLAS Error: " << __FILE__ << ":" << __LINE__      \
                      << ", Status Code: " << status << std::endl;            \
            exit(1);                                                          \
        }                                                                     \
    }

int main() {
    // Matrix dimensions
    int n = 1000; // Example size for square matrices

    // Track overall execution time
    auto total_start = std::chrono::high_resolution_clock::now();

    // Allocate and initialize host memory
    // Change Double -> Float
    std::vector<float> A(n * n, 1.0); // Matrix A
    std::vector<float> B(n * n, 1.0); // Matrix B
    std::vector<float> C(n * n, 0.0); // Result matrix

    // Allocate device memory
    double *d_A, *d_B, *d_C, *d_intermediate;
    CUDA_CHECK(hipMalloc(&d_A, n * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B, n * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, n * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_intermediate, n * n * sizeof(double)));

    // Copy matrices A and B to the device
    CUDA_CHECK(hipMemcpy(d_A, A.data(), n * n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B.data(), n * n * sizeof(double), hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Perform first matrix multiplication: d_intermediate = d_A * d_B
    double alpha = 1.0;
    double beta = 0.0;
    CUBLAS_CHECK(hipblasDgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        d_A, n,
        d_B, n,
        &beta,
        d_intermediate, n
    ));

    // Perform second matrix multiplication: d_C = d_intermediate * d_B
    CUBLAS_CHECK(hipblasDgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        d_intermediate, n,
        d_B, n,
        &beta,
        d_C, n
    ));

    // Copy final result back to host
    CUDA_CHECK(hipMemcpy(C.data(), d_C, n * n * sizeof(double), hipMemcpyDeviceToHost));

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_intermediate);
    hipblasDestroy(handle);

    // Track total execution time
    auto total_end = std::chrono::high_resolution_clock::now();
    std::cout << "Total execution time: "
              << std::chrono::duration<double>(total_end - total_start).count()
              << " seconds" << std::endl;

    return 0;
}
